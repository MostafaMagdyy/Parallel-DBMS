#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__device__ float atomicMaxFloat(float* address, float val) {
    int* address_as_int = (int*)address;
    int old = *address_as_int;
    int assumed;
    
    do {
        assumed = old;
        old = atomicCAS(address_as_int, assumed, 
                        __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    
    return __int_as_float(old);
}


// Kernel to find maximum element using warp synchronization
__global__ void findMaxElement(float* input, float* output, int size) {
    __shared__ float warp_maxes[32];
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int lane_id = threadIdx.x % 32; 
    int warp_id = threadIdx.x / 32; 
    
    float local_max = -INFINITY;
    if (tid < size) {
        local_max = input[tid];
    }
    
    // Perform warp-level reduction using shuffle operations
    for (int offset = 16; offset > 0; offset /= 2) {
        float neighbor = __shfl_down_sync(0xffffffff, local_max, offset);
        local_max = fmaxf(local_max, neighbor);
    }
    
    // First thread in each warp writes result to shared memory
    if (lane_id == 0) {
        warp_maxes[warp_id] = local_max;
    }
    
    __syncthreads();
    
    if (warp_id == 0 && lane_id < (blockDim.x + 31) / 32) {
        local_max = warp_maxes[lane_id];
        
        // Final warp reduction
        for (int offset = 16; offset > 0; offset /= 2) {
            float neighbor = __shfl_down_sync(0xffffffff, local_max, offset);
            local_max = fmaxf(local_max, neighbor);
        }
        
        // First thread in block writes result to global memory
        if (lane_id == 0) {
            atomicMaxFloat(output, local_max);
        }
    }
}

void readInputFile(const char *filename, float **data, int *size) {
    FILE *file = fopen(filename, "r");
    if (!file) {
        fprintf(stderr, "Error opening file %s\n", filename);
        exit(EXIT_FAILURE);
    }

    // Read the size of the array
    fscanf(file, "%d", size);

    // Allocate memory
    *data = (float *)malloc(*size * sizeof(float));

    // Read the array elements
    for (int i = 0; i < *size; i++) {
        fscanf(file, "%f", &(*data)[i]);
    }

    fclose(file);
    printf("Read %d elements from input file\n", *size);
}

void writeOutputFile(const char *filename, double result) {
    FILE *file = fopen(filename, "w");
    if (!file) {
        fprintf(stderr, "Error opening file %s\n", filename);
        exit(EXIT_FAILURE);
    }

    result = round(result * 1000.0) / 1000.0;

    fprintf(file, "%.3f\n", result);
    fclose(file);
}


int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s <inputfile> <outputfile>\n", argv[0]);
        return -1;
    }

    const char *inputFile = argv[1];
    const char *outputFile = argv[2];

    float *h_input;
    int size;
    readInputFile(inputFile, &h_input, &size);

    float *d_input, *d_output;
    hipMalloc(&d_input, size * sizeof(double));
    hipMalloc(&d_output, sizeof(double));

    hipMemcpy(d_input, h_input, size * sizeof(double), hipMemcpyHostToDevice);

    double neg_inf = -INFINITY;
    hipMemcpy(d_output, &neg_inf, sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;

    printf("Launching max element finder kernel with grid size %d, block size %d\n", gridSize, blockSize);
    
    findMaxElement<<<gridSize, blockSize>>>(d_input, d_output, size);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        return -1;
    }


    float h_output;
    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    writeOutputFile(outputFile, h_output);

    hipFree(d_input);
    hipFree(d_output);
    free(h_input);

    return 0;
}