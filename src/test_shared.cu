#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int device = 0, val = 0;
    hipSetDevice(device);

    hipDeviceGetAttribute(&val,
        hipDeviceAttributeMaxSharedMemoryPerBlock,
        device);
    std::cout << "Max shared memory per block: "
              << (val/1024) << " KB\n";

    hipDeviceGetAttribute(&val,
        hipDeviceAttributeMaxSharedMemoryPerMultiprocessor,
        device);
    std::cout << "Max shared memory per SM:    "
              << (val/1024) << " KB\n";
    return 0;
}
