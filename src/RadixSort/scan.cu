#include "hip/hip_runtime.h"
#include "scan.h"

#define MAX_BLOCK_SZ 128
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5

// #define ZERO_BANK_CONFLICTS

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) \
    ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif

__global__ void gpu_add_block_sums(uint64_t *const d_out,
                                   const uint64_t *const d_in,
                                   uint64_t *const d_block_sums,
                                   const size_t numElems)
{
    // uint64_t glbl_t_idx = blockDim.x * blockIdx.x + threadIdx.x;
    uint64_t d_block_sum_val = d_block_sums[blockIdx.x];

    // uint64_t d_in_val_0 = 0;
    // uint64_t d_in_val_1 = 0;

    // Simple implementation's performance is not significantly (if at all)
    //  better than previous verbose implementation
    uint64_t cpy_idx = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    if (cpy_idx < numElems)
    {
        d_out[cpy_idx] = d_in[cpy_idx] + d_block_sum_val;
        if (cpy_idx + blockDim.x < numElems)
            d_out[cpy_idx + blockDim.x] = d_in[cpy_idx + blockDim.x] + d_block_sum_val;
    }
}

// Modified version of Mark Harris' implementation of the Blelloch scan
//  according to https://www.mimuw.edu.pl/~ps209291/kgkp/slides/scan.pdf
__global__ void gpu_prescan(uint64_t *const d_out,
                            const uint64_t *const d_in,
                            uint64_t *const d_block_sums,
                            const uint64_t len,
                            const uint64_t shmem_sz,
                            const uint64_t max_elems_per_block)
{
    // Allocated on invocation
    extern __shared__ uint64_t s_out[];

    int thid = threadIdx.x;
    int ai = thid;
    int bi = thid + blockDim.x;

    // Zero out the shared memory
    // Helpful especially when input size is not power of two
    s_out[thid] = 0;
    s_out[thid + blockDim.x] = 0;
    // If CONFLICT_FREE_OFFSET is used, shared memory size
    //  must be a 2 * blockDim.x + blockDim.x/num_banks
    s_out[thid + blockDim.x + (blockDim.x >> LOG_NUM_BANKS)] = 0;

    __syncthreads();

    // Copy d_in to shared memory
    // Note that d_in's elements are scattered into shared memory
    //  in light of avoiding bank conflicts
    uint64_t cpy_idx = max_elems_per_block * blockIdx.x + threadIdx.x;
    if (cpy_idx < len)
    {
        s_out[ai + CONFLICT_FREE_OFFSET(ai)] = d_in[cpy_idx];
        if (cpy_idx + blockDim.x < len)
            s_out[bi + CONFLICT_FREE_OFFSET(bi)] = d_in[cpy_idx + blockDim.x];
    }

    // For both upsweep and downsweep:
    // Sequential indices with conflict free padding
    //  Amount of padding = target index / num banks
    //  This "shifts" the target indices by one every multiple
    //   of the num banks
    // offset controls the stride and starting index of
    //  target elems at every iteration
    // d just controls which threads are active
    // Sweeps are pivoted on the last element of shared memory

    // Upsweep/Reduce step
    int offset = 1;
    for (int d = max_elems_per_block >> 1; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)
        {
            int ai = offset * ((thid << 1) + 1) - 1;
            int bi = offset * ((thid << 1) + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            s_out[bi] += s_out[ai];
        }
        offset <<= 1;
    }

    // Save the total sum on the global block sums array
    // Then clear the last element on the shared memory
    if (thid == 0)
    {
        d_block_sums[blockIdx.x] = s_out[max_elems_per_block - 1 + CONFLICT_FREE_OFFSET(max_elems_per_block - 1)];
        s_out[max_elems_per_block - 1 + CONFLICT_FREE_OFFSET(max_elems_per_block - 1)] = 0;
    }

    // Downsweep step
    for (int d = 1; d < max_elems_per_block; d <<= 1)
    {
        offset >>= 1;
        __syncthreads();

        if (thid < d)
        {
            int ai = offset * ((thid << 1) + 1) - 1;
            int bi = offset * ((thid << 1) + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            uint64_t temp = s_out[ai];
            s_out[ai] = s_out[bi];
            s_out[bi] += temp;
        }
    }
    __syncthreads();

    // Copy contents of shared memory to global memory
    if (cpy_idx < len)
    {
        d_out[cpy_idx] = s_out[ai + CONFLICT_FREE_OFFSET(ai)];
        if (cpy_idx + blockDim.x < len)
            d_out[cpy_idx + blockDim.x] = s_out[bi + CONFLICT_FREE_OFFSET(bi)];
    }
}

void sum_scan_blelloch(uint64_t *const d_out,
                       const uint64_t *const d_in,
                       const size_t numElems)
{
    // Zero out d_out
    checkCudaErrors(hipMemset(d_out, 0, numElems * sizeof(uint64_t)));

    // Set up number of threads and blocks

    uint64_t block_sz = MAX_BLOCK_SZ / 2;
    uint64_t max_elems_per_block = 2 * block_sz; // due to binary tree nature of algorithm

    // If input size is not power of two, the remainder will still need a whole block
    // Thus, number of blocks must be the ceiling of input size / max elems that a block can handle
    // uint64_t grid_sz = (uint64_t) std::ceil((double) numElems / (double) max_elems_per_block);
    // UPDATE: Instead of using ceiling and risking miscalculation due to precision, just automatically
    //  add 1 to the grid size when the input size cannot be divided cleanly by the block's capacity
    uint64_t grid_sz = numElems / max_elems_per_block;
    // Take advantage of the fact that integer division drops the decimals
    if (numElems % max_elems_per_block != 0)
        grid_sz += 1;

    // Conflict free padding requires that shared memory be more than 2 * block_sz
    uint64_t shmem_sz = max_elems_per_block + ((max_elems_per_block) >> LOG_NUM_BANKS);

    // Allocate memory for array of total sums produced by each block
    // Array length must be the same as number of blocks
    uint64_t *d_block_sums;
    checkCudaErrors(hipMalloc(&d_block_sums, sizeof(uint64_t) * grid_sz));
    checkCudaErrors(hipMemset(d_block_sums, 0, sizeof(uint64_t) * grid_sz));

    // Sum scan data allocated to each block
    // gpu_sum_scan_blelloch<<<grid_sz, block_sz, sizeof(uint64_t) * max_elems_per_block >>>(d_out, d_in, d_block_sums, numElems);
    gpu_prescan<<<grid_sz, block_sz, sizeof(uint64_t) * shmem_sz>>>(d_out,
                                                                    d_in,
                                                                    d_block_sums,
                                                                    numElems,
                                                                    shmem_sz,
                                                                    max_elems_per_block);

    // Sum scan total sums produced by each block
    // Use basic implementation if number of total sums is <= 2 * block_sz
    //  (This requires only one block to do the scan)
    if (grid_sz <= max_elems_per_block)
    {
        uint64_t *d_dummy_blocks_sums;
        checkCudaErrors(hipMalloc(&d_dummy_blocks_sums, sizeof(uint64_t)));
        checkCudaErrors(hipMemset(d_dummy_blocks_sums, 0, sizeof(uint64_t)));
        // gpu_sum_scan_blelloch<<<1, block_sz, sizeof(uint64_t) * max_elems_per_block>>>(d_block_sums, d_block_sums, d_dummy_blocks_sums, grid_sz);
        gpu_prescan<<<1, block_sz, sizeof(uint64_t) * shmem_sz>>>(d_block_sums,
                                                                  d_block_sums,
                                                                  d_dummy_blocks_sums,
                                                                  grid_sz,
                                                                  shmem_sz,
                                                                  max_elems_per_block);
        
        checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipFree(d_dummy_blocks_sums));
    }
    // Else, recurse on this same function as you'll need the full-blown scan
    //  for the block sums
    else
    {
        uint64_t *d_in_block_sums;
        checkCudaErrors(hipMalloc(&d_in_block_sums, sizeof(uint64_t) * grid_sz));
        checkCudaErrors(hipMemcpy(d_in_block_sums, d_block_sums, sizeof(uint64_t) * grid_sz, hipMemcpyDeviceToDevice));
        sum_scan_blelloch(d_block_sums, d_in_block_sums, grid_sz);
        checkCudaErrors(hipFree(d_in_block_sums));
    }
    gpu_add_block_sums<<<grid_sz, block_sz>>>(d_out, d_out, d_block_sums, numElems);

    checkCudaErrors(hipFree(d_block_sums));
}
